#include "hip/hip_runtime.h"
#include <sstream>
#include <iostream>
#include <hip/hip_runtime.h>
#include "hip/hip_runtime_api.h"
#include "helper_functions.h"

__global__ void kernel
        (double *vec, double scalar, unsigned int num_elements)
{
    unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < num_elements) {
        vec[idx] = vec[idx] * scalar;
    }
}

void run_kernel
(double *vec, double scalar, unsigned int num_elements)
{
    size_t byte_size = num_elements * sizeof(double);

    double *gpu_ptr;
    checkCudaErrors(hipMalloc((void**)&gpu_ptr, byte_size));
    checkCudaErrors(hipMemcpy(gpu_ptr, vec, byte_size, hipMemcpyHostToDevice));
    dim3 dimBlock(256, 1, 1);
    dim3 dimGrid(ceil((double)num_elements / dimBlock.x));
    kernel <<<dimGrid, dimBlock>>>
        (gpu_ptr, scalar, num_elements);
    checkCudaErrors(hipGetLastError());
    checkCudaErrors(hipDeviceSynchronize());
    checkCudaErrors(hipMemcpy(vec, gpu_ptr, byte_size, hipMemcpyDeviceToHost));
    checkCudaErrors(hipFree(gpu_ptr));
}